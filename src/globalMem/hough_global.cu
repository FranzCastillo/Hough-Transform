#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <fstream>
#include <algorithm>
#include <opencv2/opencv.hpp>
#include "../common/pgm.h"

const double degreeInc = 0.5; // use 4.0 for reinforced-lines image and 0.5 for original
const int degreeBins = static_cast<int>(180.0 / degreeInc);
const int rBins = 100;
const double radInc = degreeInc * M_PI / 180.0;


//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc)
{
    double rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2.0;
    *acc = new int[rBins * degreeBins];
    memset(*acc, 0, sizeof(int) * rBins * degreeBins);
    int xCent = w / 2;
    int yCent = h / 2;
    double rScale = 2.0 * rMax / rBins;

    for (int i = 0; i < w; i++)
    {
        for (int j = 0; j < h; j++)
        {
            int idx = j * w + i;
            if (pic[idx] > 0)
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;
                double theta = 0.0;
                for (int tIdx = 0; tIdx < degreeBins; tIdx++)
                {
                    double r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    if (rIdx >= 0 && rIdx < rBins)
                    {
                        (*acc)[rIdx * degreeBins + tIdx]++;
                    }
                    theta += radInc;
                }
            }
        }
    }
}

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc,
    double rMax, double rScale, double *d_Cos, double *d_Sin)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID >= w * h)
    return;

  int xCent = w / 2;
  int yCent = h / 2;
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
        double r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
        int rIdx = (r + rMax) / rScale;
        if (rIdx >= 0 && rIdx < rBins)
        {
            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
  }
}

//*****************************************************************
int main(int argc, char **argv)
{
  int i;

  PGMImage inImg(argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;
  
  double* d_Cos;
  double* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (double) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (double) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  double *pcCos = (double *)malloc(sizeof(double) * degreeBins);
  double *pcSin = (double *)malloc(sizeof(double) * degreeBins);
  double rad = 0.0;
  for (int i = 0; i < degreeBins; i++)
  {
      pcCos[i] = cos(rad);
      pcSin[i] = sin(rad);
      rad += radInc;
  }

  hipMemcpy(d_Cos, pcCos, sizeof(double) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof(double) * degreeBins, hipMemcpyHostToDevice);

  double rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
  double rScale = 2 * rMax / rBins;

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

  hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
  hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
  hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  // 1 thread por pixel
  int blockNum = ceil(w * h / 256);

  // Crear eventos CUDA
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Registrar el tiempo de inicio
  hipEventRecord(start);

  GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  // Registrar el tiempo de fin
  hipEventRecord(stop);
  // Esperar a que el evento stop se complete
  hipEventSynchronize(stop);

  // Calcular el tiempo transcurrido
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Tiempo de ejecución del kernel: %f ms\n", milliseconds);

  // get results from device
  hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");


  cv::Mat originalImage = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
  if (originalImage.empty())
  {
      std::cerr << "Error: Could not load image " << argv[1] << std::endl;
      return -1;
  }

  // Convert to BGR color space to draw colored lines
  cv::Mat colorImage;
  cv::cvtColor(originalImage, colorImage, cv::COLOR_GRAY2BGR);

  // Set threshold for significant lines

  const int threshold = 4200; //Use 2500 for reinforced-lines image an 4200 for original lines

  // Apply threshold
  for (int rIdx = 0; rIdx < rBins; rIdx++)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            int idx = rIdx * degreeBins + tIdx;
            if (h_hough[idx] >= threshold)
            {
                float theta = tIdx * radInc;
                float r = rIdx * rScale - rMax;

                double cosTheta = cos(theta);
                double sinTheta = sin(theta);

                // Points where the line crosses the borders of the image
                cv::Point pt1, pt2;

                // Since sinTheta and cosTheta can be zero, we need to handle those cases
                if (fabs(sinTheta) > 1e-6)
                {
                    // Compute the intersection with the left and right borders
                    pt1.x = 0;
                    pt1.y = (r - (pt1.x - w / 2) * cosTheta) / sinTheta + h / 2;

                    pt2.x = w;
                    pt2.y = (r - (pt2.x - w / 2) * cosTheta) / sinTheta + h / 2;
                }
                else
                {
                    // sinTheta is zero, line is horizontal
                    pt1.y = 0;
                    pt1.x = (r - (pt1.y - h / 2) * sinTheta) / cosTheta + w / 2;

                    pt2.y = h;
                    pt2.x = (r - (pt2.y - h / 2) * sinTheta) / cosTheta + w / 2;
                }

                // Adjust y-coordinates to account for the inverted y-axis
                pt1.y = h - pt1.y;
                pt2.y = h - pt2.y;

                // Draw the line on the image
                cv::line(colorImage, pt1, pt2, cv::Scalar(0, 255, 0), 1);
            }
        }
    }

  cv::imwrite("output.png", colorImage);

  // cleanup
  free(cpuht);
  free(h_hough);
  free(pcCos);
  free(pcSin);
  hipFree(d_in);
  hipFree(d_hough);
  hipFree(d_Cos);
  hipFree(d_Sin);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
